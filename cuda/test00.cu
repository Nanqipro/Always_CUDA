
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void hello_from_gpu(){
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("Hello world from block %d and thread %d \n",bid,tid);
    // printf("Hello world from thr GPU\n");
}
 
int main(void){
    hello_from_gpu<<<2,4>>>();
    hipDeviceSynchronize();
 
    return 0;
}