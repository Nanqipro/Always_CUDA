
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// 错误检查宏
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const int DSIZE = 4096;
const int block_size = 256;

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_a, *d_b, *d_c;

    h_A = (float*)malloc(DSIZE * sizeof(float));
    h_B = (float*)malloc(DSIZE * sizeof(float));
    h_C = (float*)malloc(DSIZE * sizeof(float));

    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
        h_C[i] = 0;
    }

    hipMalloc((void**)&d_a, DSIZE * sizeof(float));
    hipMalloc((void**)&d_b, DSIZE * sizeof(float));
    hipMalloc((void**)&d_c, DSIZE * sizeof(float));

    cudaCheckErrors("hipMalloc failed");

    hipMemcpy(d_a, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_B, DSIZE * sizeof(float), hipMemcpyHostToDevice);

    cudaCheckErrors("hipMemcpy failed");

    vector_add<<<DSIZE / block_size, block_size>>>(d_a, d_b, d_c, DSIZE);

    cudaCheckErrors("kernel launch failed");

    hipMemcpy(h_C, d_c, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    cudaCheckErrors("hipMemcpy failed");

    for (int i = 0; i < DSIZE; i++) {
        printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
