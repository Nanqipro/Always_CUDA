#include "hip/hip_runtime.h"
#include <stdio.h>
#include "error.cuh"
#include <hip/hip_runtime.h>
// 矩阵乘法的朴素实现
#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

// 计算矩阵乘法：C = A * B，矩阵A的维度为M*K，矩阵B的维度为K*N

#define M 512
#define K 512
#define N 512

const int NUM_REPEATS = 10;

void initial(real *array, int size){
    for(int i = 0; i < size; i++){
        array[i] = (real)(rand() % 10 + 1);
    }
}


void print_matrix(real *C, int m,int n){
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }
}

__global__ void matrix_mul_01(real *A, real *B,real *C,int m, int k, int n){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if(ix < m && iy < n){
        real Cvalue = 0;
        for(int kk = 0; kk < k; kk++){
            Cvalue += A[ix * k + kk] * B[kk * n + iy]; //行优先矩阵
        }
        C[ix * n + iy] = Cvalue;
    }
}


void timing(real *d_A, real *d_B, real *d_C, int m, int k, int n){
    int dimx = 2;
    int dimy = 2;

    dim3 dimBlock(dimx,dimy);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);

    float t1_sum = 0.0;
    for(int i = 0; i < NUM_REPEATS; i++){
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start, 0));
        hipEventQuery(start);
        matrix_mul_01<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,M,K,N);
        CHECK(hipEventRecord(stop, 0));
        CHECK(hipEventSynchronize(stop));
        float t1;
        CHECK(hipEventElapsedTime(&t1, start, stop));
        printf("%f\n",t1);
        if(i>0){
            t1_sum += t1;
        }
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));

    }
    printf("Average time: %f\n",t1_sum/(NUM_REPEATS-1));
}

int main(int argc, char **argv){
    int size_A = M * K;
    int size_B = K * N;
    int size_C = M * N;
    real *A, *B, *C;
    A = (real*)malloc(size_A * sizeof(real));
    B = (real*)malloc(size_B * sizeof(real));
    C = (real*)malloc(size_C * sizeof(real));

    initial(A, size_A);
    initial(B, size_B);

    // print_matrix(A,M,K);
    // print_matrix(B,K,N);
    
    real *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A * sizeof(real));  
    hipMalloc((void**)&d_B, size_B * sizeof(real));
    hipMalloc((void**)&d_C, size_C * sizeof(real));

    hipMemcpy(d_A, A, size_A * sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B * sizeof(real), hipMemcpyHostToDevice);

    // int dimx = 2;
    // int dimy = 2;

    // dim3 dimBlock(dimx,dimy);
    // dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
    // matrix_mul_01<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,M,K,N);

    // print_matrix(C,M,N);

    timing(d_A, d_B, d_C, M, K, N);

    hipMemcpy(C, d_C, size_C * sizeof(real), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    return 0;
}

// #include <stdio.h>
// #include "error.cuh"
// #include <hip/hip_runtime.h>

// // 矩阵乘法的朴素实现
// // 计算矩阵乘法：C = A * B，矩阵A的维度为M*K，矩阵B的维度为K*N

// #define M 512
// #define K 512
// #define N 512

// const int NUM_REPEATS = 10;

// void initial(float *array, int size){
//     for(int i = 0; i < size; i++){
//         array[i] = (float)(rand() % 10 + 1);
//     }
// }

// void print_matrix(float *C,int m,int n){
//     for (int i = 0; i < M; i++) {
//         for (int j = 0; j < N; j++) {
//             printf("%f ", C[i * N + j]);
//         }
//         printf("\n");
//     }
// }

// __global__ void matrix_mul_01(float *A, float *B, float *C, int m, int k, int n){
//     int ix = blockIdx.x * blockDim.x + threadIdx.x;
//     int iy = blockIdx.y * blockDim.y + threadIdx.y;
//     if(ix < m && iy < n){
//         float Cvalue = 0;
//         for(int kk = 0; kk < k; kk++){
//             Cvalue += A[ix * k + kk] * B[kk * n + iy]; //行优先矩阵
//         }
//         C[ix * n + iy] = Cvalue;
//     }
// }

// void timing(float *d_A, float *d_B, float *d_C, int m, int k, int n){
//     int dimx = 2;
//     int dimy = 2;

//     dim3 dimBlock(dimx,dimy);
//     dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);

//     float t1_sum = 0.0;
//     for(int i = 0; i < NUM_REPEATS; i++){
//         hipEvent_t start, stop;
//         CHECK(hipEventCreate(&start));
//         CHECK(hipEventCreate(&stop));
//         CHECK(hipEventRecord(start, 0));
//         hipEventQuery(start);
//         matrix_mul_01<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,M,K,N);
//         CHECK(hipEventRecord(stop, 0));
//         CHECK(hipEventSynchronize(stop));
//         float t1;
//         CHECK(hipEventElapsedTime(&t1, start, stop));
//         printf("%f\n",t1);
//         if(i>0){
//             t1_sum += t1;
//         }
//         CHECK(hipEventDestroy(start));
//         CHECK(hipEventDestroy(stop));
//     }
//     printf("Average time: %f\n",t1_sum/(NUM_REPEATS-1));
// }

// int main(int argc, char **argv){
//     int size_A = M * K;
//     int size_B = K * N;
//     int size_C = M * N;
//     float *A, *B, *C;
//     A = (float*)malloc(size_A * sizeof(float));
//     B = (float*)malloc(size_B * sizeof(float));
//     C = (float*)malloc(size_C * sizeof(float));

//     initial(A, size_A);
//     initial(B, size_B);

//     print_matrix(A,M,K);
//     print_matrix(B,K,N);
    
//     float *d_A, *d_B, *d_C;
//     hipMalloc((void**)&d_A, size_A * sizeof(float));  
//     hipMalloc((void**)&d_B, size_B * sizeof(float));
//     hipMalloc((void**)&d_C, size_C * sizeof(float));

//     hipMemcpy(d_A, A, size_A * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_B, B, size_B * sizeof(float), hipMemcpyHostToDevice);

//     int dimx = 2;
//     int dimy = 2;

//     dim3 dimBlock(dimx,dimy);
//     dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
//     matrix_mul_01<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,M,K,N);

//     print_matrix(C,M,N);

//     timing(d_A, d_B, d_C, M, K, N);

//     hipMemcpy(C, d_C, size_C * sizeof(float), hipMemcpyDeviceToHost);
//     hipFree(d_A);
//     hipFree(d_B);
//     hipFree(d_C);
//     free(A);
//     free(B);
//     free(C);
//     return 0;
// }
