#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

// 不用GPU的矩阵乘法
void matrix_mul_naive(float *A, float *B, float *C, int m, int k, int n){
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++){
            float sum = 0;
            for(int l = 0; l < k; l++){
                sum += A[i * k + l] * B[l * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// 比较两种算法的差异，用于判断是否正确
float compare(float *hostC, float *serialC, int M, int N)
{
    float error = 0;
    for (int i = 0; i < M * N; i++)
    {
        error = fmax(error, fabs(hostC[i] - serialC[i]));
    }
    return error;
}

// 用GPU进行矩阵乘法v1

__global__ void matrix_mul_01(float *A, float *B, float *C, int m, int k, int n){
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    if(row < m && col < n){
        float sum = 0;
        for(int l = 0; l < k; l++){
            sum += A[row * k + l] * B[l * n + col];
        }
        C[row * n + col] =sum;
    }
}

// 用GPU进行矩阵乘法v2
template <int BLOCK_DIM> //模板函数
__global__ void matrix_mul_02(float *dA, float *dB, float *dC, int M, int K, int N)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    float tmp = 0.0f;
    __shared__ float SA[BLOCK_DIM][BLOCK_DIM];
    __shared__ float SB[BLOCK_DIM][BLOCK_DIM];
    int width = (K + BLOCK_DIM - 1) / BLOCK_DIM;
    for (int ph = 0; ph < width; ph++)
    {
        if (row < M && threadIdx.y + ph * BLOCK_DIM < K)
        {
            SA[threadIdx.x][threadIdx.y] = dA[row * K + threadIdx.y + ph * BLOCK_DIM];
        }
        else
        {
            SA[threadIdx.x][threadIdx.y] = 0.0f;
        }
        if (threadIdx.x + ph * BLOCK_DIM < K && col < N)
        {
            SB[threadIdx.x][threadIdx.y] = dB[(threadIdx.x + ph * BLOCK_DIM) * N + col];
        }
        else
        {
            SB[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int s = 0; s < BLOCK_DIM; s++)
        {
            tmp += SA[threadIdx.x][s] * SB[s][threadIdx.y];
        }
        __syncthreads();
    }
    if (row < M && col < N)
    {
        dC[row * N + col] = tmp;
    }
}

void hostMatrix(float *A, float *B, float *C, int m, int k, int n){
    double start = get_walltime();
    double elapsed = 0;

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, m * k * sizeof(float));
    hipMalloc((void **)&d_B, k * n * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);

    int BLOCK_DIM_x = 32;
    int BLOCK_DIM_y = 32;
    int num_blocks_x = (m + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
    int num_blocks_y = (n + BLOCK_DIM_y - 1) / BLOCK_DIM_y;

    dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y,1);
    dim3 grid_dim(num_blocks_x, num_blocks_y,1);

    int NUM_REPEATS = 100;
    matrix_mul_01<<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    // matrix_mul_02<32><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    hipEvent_t start_event, stop_event;
    float kernel_time = 0;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event,0);
    
    for(int i = 0; i < NUM_REPEATS; i++){
        matrix_mul_01<<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
        // matrix_mul_02<32><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(stop_event,0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);

    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    elapsed = get_walltime() - start;
    printf("M-K-N: %d-%d-%d\n", m, k, n);
    printf("GPU use time: %.4f second\n", elapsed);
    printf("kernel time: %.4f second, %.4f ms\n", kernel_time / (NUM_REPEATS * 1000.), kernel_time / NUM_REPEATS);
    printf("grid dim: %d, %d, %d\n", grid_dim.x, grid_dim.y, grid_dim.z);
    printf("block dim: %d, %d, %d\n", block_dim.x, block_dim.y, block_dim.z);
}

int main(int argc, char **argv){
    float *A, *B, *C, *serialC;
    int M = 1024;
    int K = 1024;
    int N = 1024;
    A = (float *)malloc(M * K * sizeof(float));
    B = (float *)malloc(K * N * sizeof(float));
    C = (float *)malloc(M * N * sizeof(float));
    serialC = (float *)malloc(M * N * sizeof(float));
    //初始化
    for (int i = 0; i < M * K; i++)
    {
        A[i] = i%3;
    }
    for (int i = 0; i < K * N; i++)
    {
        B[i] = i%3;
    }
    //用GPU进行矩阵乘法
    hostMatrix(A, B, C, M, K, N);

    double start = get_walltime();
    double elapsed = 0;
    //不用GPU进行矩阵乘法
    matrix_mul_naive(A, B, serialC, M, K, N);

    elapsed = get_walltime() - start;
    //比较计算是否正确
    float error = compare(C, serialC, M, N);
    printf("CPU use time: %.4f second\n", elapsed);
    printf("error: %.4f\n", error);
    free(A);
    free(B);
    free(C);
    free(serialC);
    return 0;

}


