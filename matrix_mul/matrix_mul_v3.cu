#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
const int TM = 4;
const int TN = 4;
const int BLOCK_DIM_x = 32;
const int BLOCK_DIM_y = 32;
const int BM = TM * BLOCK_DIM_x;
const int BN = TN * BLOCK_DIM_y;
const int BK = 8;
double
get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
// 不用GPU的矩阵乘法
void matrix_mul_naive(float *A, float *B, float *C, int m, int k, int n){
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++){
            float sum = 0;
            for(int l = 0; l < k; l++){
                sum += A[i * k + l] * B[l * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}
// 比较两种算法的差异，用于判断是否正确
float compare(float *hostC, float *serialC, int M, int N)
{
    float error = 0;
    for (int i = 0; i < M * N; i++)
    {
        error = fmax(error, fabs(hostC[i] - serialC[i]));
    }
    return error;
}


// 用GPU进行矩阵乘法v3--瓦片化（tiling）方法
template<int BM,int BN,int BK,int TM,int TN>
__global__ void matrix_mul_03(float *A, float *B, float *C, int m, int k, int n){
    __shared__ float s_A[BM*BK];
    __shared__ float s_B[BK*BN];
    //indA 和 indB 分别表示矩阵A和B中当前线程块开始处理的位置。
    int indA = TM * (threadIdx.x+blockIdx.x*blockDim.x);
    int indB = TN * (threadIdx.y+blockIdx.y*blockDim.y);
    int width = (k+BK-1)/BK;//处理的总瓦片数
    float tmp[TM*TN] = {0.0f};//初始化

    for(int i=0;i<width;i++){
        //把A加载到共享内存
        for(int index_q = 0;index_q < TM ;index_q++){
            for(int index_k = 0;index_k < BK;index_k++){
                if(index_q+indA < m && index_k+i*BK < k) {
                    s_A[(threadIdx.x* TM + index_q)*BK + index_k] = A[(index_q+indA)*k + index_k+i*BK];
                }
                else{
                    s_A[(threadIdx.x* TM + index_q)*BK + index_k] = 0.0f;
                } 
            }
        }
        __syncthreads();
        //把B加载到共享内存
        for(int index_v = 0;index_v < TN ;index_v++){
            for(int index_k = 0;index_k < BK;index_k++){
                if(index_v+indB < n && index_k+i*BK < k) {
                    s_B[(threadIdx.y* TN + index_v)*BK + index_k] = B[(index_v+indB)*k + index_k+i*BK];
                }
                else{
                    s_B[(threadIdx.y* TN + index_v)*BK + index_k] = 0.0f;
                }
            }
        }
        __syncthreads();
        //瓦片的计算
        for(int i = 0;i < TM;i++){
            for(int j = 0;j < TN;j++){
                for(int k = 0;k < BK;k++){
                    tmp[i*TN+j] += s_A[(threadIdx.x*TM+i)*BK+k] * s_B[k * BN + threadIdx.y * TN + j];
                }
            }
        }
        __syncthreads();
    }
    //存储结果
    for(int i = 0;i < TM;i++){
        for(int j = 0;j < TN;j++){
            if(indA+i < m && indB+j < n){
                C[(indA+i)*n + indB+j] = tmp[i*TN+j];
            }
        }
    }
}



void hostMatrix(float *A, float *B, float *C, int m, int k, int n){
    double start = get_walltime();
    double elapsed = 0;

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, m * k * sizeof(float));
    hipMalloc((void **)&d_B, k * n * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);

    int BLOCK_DIM_x = 32;
    int BLOCK_DIM_y = 32;
    int num_blocks_x = (m + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
    int num_blocks_y = (n + BLOCK_DIM_y - 1) / BLOCK_DIM_y;

    dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y,1);
    dim3 grid_dim(num_blocks_x, num_blocks_y,1);

    int NUM_REPEATS = 100;
    // matrix_mul_01<<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    // matrix_mul_02<32><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    matrix_mul_03<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    hipEvent_t start_event, stop_event;
    float kernel_time = 0;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event,0);
    
    for(int i = 0; i < NUM_REPEATS; i++){
        // matrix_mul_01<<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
        // matrix_mul_02<32><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
        matrix_mul_03<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(stop_event,0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);

    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    elapsed = get_walltime() - start;
    printf("M-K-N: %d-%d-%d\n", m, k, n);
    printf("GPU use time: %.4f second\n", elapsed);
    printf("kernel time: %.4f second, %.4f ms\n", kernel_time / (NUM_REPEATS * 1000.), kernel_time / NUM_REPEATS);
    printf("grid dim: %d, %d, %d\n", grid_dim.x, grid_dim.y, grid_dim.z);
    printf("block dim: %d, %d, %d\n", block_dim.x, block_dim.y, block_dim.z);
}

int main(int argc, char **argv){
    float *A, *B, *C, *serialC;
    int M = 1024;
    int K = 1024;
    int N = 1024;
    A = (float *)malloc(M * K * sizeof(float));
    B = (float *)malloc(K * N * sizeof(float));
    C = (float *)malloc(M * N * sizeof(float));
    serialC = (float *)malloc(M * N * sizeof(float));
    //初始化
    for (int i = 0; i < M * K; i++)
    {
        A[i] = i%3;
    }
    for (int i = 0; i < K * N; i++)
    {
        B[i] = i%3;
    }
    //用GPU进行矩阵乘法
    hostMatrix(A, B, C, M, K, N);

    double start = get_walltime();
    double elapsed = 0;
    //不用GPU进行矩阵乘法
    matrix_mul_naive(A, B, serialC, M, K, N);

    elapsed = get_walltime() - start;
    //比较计算是否正确
    float error = compare(C, serialC, M, N);
    printf("CPU use time: %.4f second\n", elapsed);
    printf("error: %.4f\n", error);
    free(A);
    free(B);
    free(C);
    free(serialC);
    return 0;

}
