// #include <cuda.h>
// #include <sys/time.h>
// #include <stdio.h>
// const int TM = 4;
// const int TN = 4;
// const int BLOCK_DIM_x = 32;
// const int BLOCK_DIM_y = 32;
// const int BM = TM * BLOCK_DIM_x;
// const int BN = TN * BLOCK_DIM_y;
// const int BK = 8;
// double
// get_walltime()
// {
//     struct timeval tp;
//     gettimeofday(&tp, NULL);
//     return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
// }
// // 不用GPU的矩阵乘法
// void matrix_mul_naive(float *A, float *B, float *C, int m, int k, int n){
//     for(int i = 0; i < m; i++){
//         for(int j = 0; j < n; j++){
//             float sum = 0;
//             for(int l = 0; l < k; l++){
//                 sum += A[i * k + l] * B[l * n + j];
//             }
//             C[i * n + j] = sum;
//         }
//     }
// }
// // 比较两种算法的差异，用于判断是否正确
// void compare(float *hostC, float *serialC, int M, int N)
// {
//     float error = 0;
//     float err = 0;
//     bool tmp = true;
//     for (int i = 0; i < M * N; i++)
//     {
//         // error = fmax(error, fabs(hostC[i] - serialC[i]));
//         err = fabs(hostC[i] - serialC[i]);
//         if (err > 1e-5)
//         {
//             tmp = false;
//             printf("error:hostC[%d] = %.3f, serialC[%d] = %.3f\n", i, hostC[i], i, serialC[i]);
//             // break;
//         }
//     }
//     if (tmp)
//     {
//         printf("GPU output all right\n");
//     }
// }


// // 用GPU进行矩阵乘法v3--瓦片化（tiling）方法
// template<int BM,int BN,int BK,int TM,int TN>
// __global__ void matrix_mul_03(float *A, float *B, float *C, int m, int k, int n){
//     __shared__ float s_A[BM*BK];
//     __shared__ float s_B[BK*BN];
//     //indA 和 indB 分别表示矩阵A和B中当前线程块开始处理的位置。
//     int indA = TM * (threadIdx.x+blockIdx.x*blockDim.x);
//     int indB = TN * (threadIdx.y+blockIdx.y*blockDim.y);
//     int width = (k+BK-1)/BK;//处理的总瓦片数
//     float tmp[TM*TN] = {0.0f};//初始化

//     for(int i=0;i<width;i++){
//         //把A加载到共享内存
//         for(int index_q = 0;index_q < TM ;index_q++){
//             for(int index_k = 0;index_k < BK;index_k++){
//                 if(index_q+indA < m && index_k+i*BK < k) {
//                     s_A[(threadIdx.x* TM + index_q)*BK + index_k] = A[(index_q+indA)*k + index_k+i*BK];
//                 }
//                 else{
//                     s_A[(threadIdx.x* TM + index_q)*BK + index_k] = 0.0f;
//                 } 
//             }
//         }
//         __syncthreads();
//         //把B加载到共享内存
//         for(int index_v = 0;index_v < TN ;index_v++){
//             for(int index_k = 0;index_k < BK;index_k++){
//                 if(index_v+indB < n && index_k+i*BK < k) {
//                     s_B[(threadIdx.y* TN + index_v)*BK + index_k] = B[(index_v+indB)*k + index_k+i*BK];
//                 }
//                 else{
//                     s_B[(threadIdx.y* TN + index_v)*BK + index_k] = 0.0f;
//                 }
//             }
//         }
//         __syncthreads();
//         //瓦片的计算
//         for(int i = 0;i < TM;i++){
//             for(int j = 0;j < TN;j++){
//                 for(int k = 0;k < BK;k++){
//                     tmp[i*TN+j] += s_A[(threadIdx.x*TM+i)*BK+k] * s_B[k * BN + threadIdx.y * TN + j];
//                 }
//             }
//         }
//         __syncthreads();
//     }
//     //存储结果
//     for(int i = 0;i < TM;i++){
//         for(int j = 0;j < TN;j++){
//             if(indA+i < m && indB+j < n){
//                 C[(indA+i)*n + indB+j] = tmp[i*TN+j];
//             }
//         }
//     }
// }



// void hostMatrix(float *A, float *B, float *C, int m, int k, int n){
//     double start = get_walltime();
//     double elapsed = 0;

//     float *d_A, *d_B, *d_C;
//     cudaMalloc((void **)&d_A, m * k * sizeof(float));
//     cudaMalloc((void **)&d_B, k * n * sizeof(float));
//     cudaMalloc((void **)&d_C, m * n * sizeof(float));

//     cudaMemcpy(d_A, A, m * k * sizeof(float), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_B, B, k * n * sizeof(float), cudaMemcpyHostToDevice);


//     int num_blocks_x = (m + BM - 1) / BM;
//     int num_blocks_y = (n + BN - 1) / BN;

//     dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y,1);
//     dim3 grid_dim(num_blocks_x, num_blocks_y,1);

//     int NUM_REPEATS = 20;
//     // matrix_mul_01<<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
//     // matrix_mul_02<32><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
//     matrix_mul_03<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
//     cudaEvent_t start_event, stop_event;
//     float kernel_time = 0;
//     cudaEventCreate(&start_event);
//     cudaEventCreate(&stop_event);
//     cudaEventRecord(start_event,0);
    
//     for(int i = 0; i < NUM_REPEATS; i++){
//         // matrix_mul_01<<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
//         // matrix_mul_02<32><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
//         matrix_mul_03<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(d_A, d_B, d_C, m, k, n);
//     }
//     cudaError_t err = cudaGetLastError();
//     if (err != cudaSuccess)
//     {
//         printf("CUDA Error: %s\n", cudaGetErrorString(err));
//         // Possibly: exit(-1) if program cannot continue....
//     }

//     cudaEventRecord(stop_event,0);
//     cudaEventSynchronize(stop_event);
//     cudaEventElapsedTime(&kernel_time, start_event, stop_event);

//     cudaMemcpy(C, d_C, m * n * sizeof(float), cudaMemcpyDeviceToHost);

//     cudaFree(d_A);
//     cudaFree(d_B);
//     cudaFree(d_C);
//     cudaEventDestroy(start_event);
//     cudaEventDestroy(stop_event);
//     elapsed = get_walltime() - start;
//     printf("M-K-N: %d-%d-%d\n", m, k, n);
//     printf("GPU use time: %.4f second\n", elapsed);
//     printf("kernel time: %.4f second, %.4f ms\n", kernel_time / (NUM_REPEATS * 1000.), kernel_time / NUM_REPEATS);
//     printf("grid dim: %d, %d, %d\n", grid_dim.x, grid_dim.y, grid_dim.z);
//     printf("block dim: %d, %d, %d\n", block_dim.x, block_dim.y, block_dim.z);
// }

// int main(int argc, char **argv){
//     float *A, *B, *C, *serialC;
//     int M = 1024;
//     int K = 1024;
//     int N = 1024;
//     A = (float *)malloc(M * K * sizeof(float));
//     B = (float *)malloc(K * N * sizeof(float));
//     C = (float *)malloc(M * N * sizeof(float));
//     serialC = (float *)malloc(M * N * sizeof(float));
//     //初始化
//     for (int i = 0; i < M * K; i++)
//     {
//         A[i] = i%3;
//     }
//     for (int i = 0; i < K * N; i++)
//     {
//         B[i] = i%3;
//     }
//     //用GPU进行矩阵乘法
//     hostMatrix(A, B, C, M, K, N);

//     double start = get_walltime();
//     double elapsed = 0;
//     //不用GPU进行矩阵乘法
//     matrix_mul_naive(A, B, serialC, M, K, N);

//     elapsed = get_walltime() - start;
//     //比较计算是否正确
//     printf("CPU use time: %.4f second\n", elapsed);
//     compare(C, serialC, M, N);
//     free(A);
//     free(B);
//     free(C);
//     free(serialC);
//     return 0;

// }



#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
const int TM = 4;
const int TN = 4;
const int BLOCK_DIM_x = 32;
const int BLOCK_DIM_y = 32;
const int BM = TM * BLOCK_DIM_x;
const int BN = TN * BLOCK_DIM_y;
const int BK = 8;
double
get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
void matrixSerial(float *hostA, float *hostB, float *hostC, int M, int K, int N)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float tmp = 0;
            for (int s = 0; s < K; s++)
            {
                tmp += hostA[i * K + s] * hostB[s * N + j];
            }
            hostC[i * N + j] = tmp;
        }
    }
}
void compare(float *hostC, float *serialC, int M, int N)
{
    float error = 0;
    bool tmp = true;
    for (int i = 0; i < M * N; i++)
    {
        error = fmax(error, fabs(hostC[i] - serialC[i]));
        if (error > 1e-5)
        {
            tmp = false;
            printf("error:hostC[%d] = %.3f, serialC[%d] = %.3f\n", i, hostC[i], i, serialC[i]);
            break;
        }
    }
    if (tmp)
    {
        printf("GPU output all right\n");
    }
}

template <int BM, int BN, int BK, int TM, int TN>
__global__ void matrixKernel1st(float *dA, float *dB, float *dC, int M, int K, int N)
{
    __shared__ float SA[BM * BK];
    __shared__ float SB[BK * BN];
    int indA = TM * (threadIdx.x + blockIdx.x * blockDim.x);
    int indB = TN * (threadIdx.y + blockIdx.y * blockDim.y);
    int width = (K + BK - 1) / BK;
    float tmp[TM * TN] = {0.0f};

    for (int ph = 0; ph < width; ph++)
    {

        for (int index_q = 0; index_q < TM; index_q++)
        {
            for (int index_k = 0; index_k < BK; index_k++)
            {
                if (indA + index_q < M && index_k + ph * BK < K)
                {
                    SA[(threadIdx.x * TM + index_q) * BK + index_k] = dA[(indA + index_q) * K + index_k + ph * BK];
                }
                else
                {
                    SA[(threadIdx.x * TM + index_q) * BK + index_k] = 0.0f;
                }
            }
        }
        __syncthreads();
        for (int index_v = 0; index_v < TN; index_v++)
        {
            for (int index_k = 0; index_k < BK; index_k++)
            {

                if (indB + index_v < N && index_k + ph * BK < K)
                {

                    SB[index_k * BN + threadIdx.y * TN + index_v] = dB[(index_k + ph * BK) * N + indB + index_v];
                }
                else
                {
                    SB[index_k * BN + threadIdx.y * TN + index_v] = 0.0f;
                }
            }
        }

        __syncthreads();
        for (int index_q = 0; index_q < TM; index_q++)
        {
            for (int index_v = 0; index_v < TN; index_v++)
            {
                for (int index_k = 0; index_k < BK; index_k++)
                {
                    tmp[index_q * TN + index_v] += SA[(threadIdx.x * TM + index_q) * BK + index_k] * SB[index_k * BN + threadIdx.y * TN + index_v];
                }
            }
        }
        __syncthreads();
    }
    for (int index_q = 0; index_q < TM; index_q++)
    {
        for (int index_v = 0; index_v < TN; index_v++)
        {
            if (indA + index_q < M && indB + index_v < N)
            {
                dC[(indA + index_q) * N + indB + index_v] = tmp[index_q * TN + index_v];
            }
        }
    }
}


void hostMatrix(float *hostA, float *hostB, float *hostC, int M, int K, int N)
{
    double st, ela;
    st = get_walltime();

    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, M * K * sizeof(float));
    hipMalloc((void **)&dB, N * K * sizeof(float));
    hipMalloc((void **)&dC, M * N * sizeof(float));

    hipMemcpy(dA, hostA, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, N * K * sizeof(float), hipMemcpyHostToDevice);

    int num_blocks_x = (M + BM - 1) / BM;
    int num_blocks_y = (N + BN - 1) / BN;
    dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
    dim3 grid_dim(num_blocks_x, num_blocks_y, 1);
    int repeat = 20;
    matrixKernel1st<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    // matrixKernel2nd<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    // matrixOrigin<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < repeat; i++)
    {
        matrixKernel1st<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
        // matrixKernel2nd<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
        // matrixOrigin<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time

    hipMemcpy(hostC, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    ela = get_walltime() - st;
    printf("M-K-N: %d-%d-%d\n", M, K, N);
    printf("GPU use time: %.4f second\n", ela);
    printf("kernel time: %.4f second, %.4f ms\n", ker_time / (repeat * 1000.), ker_time / repeat);
    printf("grid dim: %d, %d, %d\n", grid_dim.x, grid_dim.y, grid_dim.z);
    printf("block dim: %d, %d, %d\n", block_dim.x, block_dim.y, block_dim.z);
}

int main()
{
    float *hostA, *hostB, *hostC, *serialC;
    int M = 1024;
    int K = 1024;
    int N = 1024;

    hostA = (float *)malloc(M * K * sizeof(float));
    hostB = (float *)malloc(N * K * sizeof(float));
    hostC = (float *)malloc(M * N * sizeof(float));
    serialC = (float *)malloc(M * N * sizeof(float));
    for (int i = 0; i < M * K; i++)
    {
        hostA[i] = i % 3;
    }
    for (int i = 0; i < N * K; i++)
    {
        hostB[i] = i % 3;
    }
    hostMatrix(hostA, hostB, hostC, M, K, N);
    double st, ela;
    st = get_walltime();
    matrixSerial(hostA, hostB, serialC, M, K, N);
    ela = get_walltime() - st;
    printf("CPU time:%.2f second\n", ela);
    compare(hostC, serialC, M, N);
    free(hostA);
    free(hostB);
    free(hostC);
    free(serialC);
    return 0;
}